#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <tuple>
#include <chrono>

#include "cuda_kernel.h"
#include "netW.hpp"
#include "utils.cuh"

using namespace std;

__global__ void layer1_conv_kernel(unsigned char *d_cuda_layer_0_output, float *d_layer_1_bias, signed char *d_cuda_layer_1_weight, float *d_cuda_layer_1_output){

    // https://github.com/ULHPC/tutorials/blob/devel/cuda/exercises/convolution/LoG_gpu_solution.cu

    int N = 32, kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.x + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < BATCH_SIZE){
            if(m < 128) {
                d_cuda_layer_1_output[index4D_cuda(b,h,w,m,32,32,128)] = d_layer_1_bias[m];
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 32) {
                for (int kW = 0; kW < kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 32) {
                        if(b < BATCH_SIZE){
                            for (int c = 0; c < 3; c++) {
                                if(m < 128) {
                                    d_cuda_layer_1_output[index4D_cuda(b,h,w,m,32,32,128)] += d_cuda_layer_1_weight[index4D_cuda(kH,kW,c,m,3,3,128)] * d_cuda_layer_0_output[index4D_cuda(b,iH,iW,c,32,32,3)];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

float layer1_conv_cuda(unsigned char x[][32][32][3], float * cuda_layer_1_output){
    
    setUniGPU();// use the second GPU on Uni-server because the first is used most of the time

    // initialize layer_0_output where x is the input image
    unsigned char (*layer_0_output)[32][32][3] = (unsigned char (*)[32][32][3]) x;

    // flatten 3D -> 1D arrays
    // flatten layer_1_weight
    signed char *cuda_layer_1_weight = (signed char *) layer_1_weight;

    // flatten layer_0_output
    unsigned char *cuda_layer_0_output = (unsigned char *) layer_0_output;

    // prepare for kernel call
    // declare storage on device
    unsigned char *d_cuda_layer_0_output; // storage on device for cuda_layer_0_output
    float *d_layer_1_bias; // storage on device for layer_1_bias
    signed char *d_cuda_layer_1_weight; // storage on device for cuda_layer_1_weight
    float *d_cuda_layer_1_output; // RESULT storage on device for cuda_layer_1_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_0_output, BATCH_SIZE*32*32*3*sizeof(unsigned char)); // dim of cuda_layer_0_output
    hipMalloc((void **) &d_layer_1_bias, 128*sizeof(float)); // 128 = dim of layer_1_bias
    hipMalloc((void **) &d_cuda_layer_1_weight, 3*3*3*128*sizeof(signed char)); // dim of layer_1_weight
    hipMalloc((void **) &d_cuda_layer_1_output, BATCH_SIZE*32*32*128*sizeof(float)); // 50176 = 28x28x64 dim of layer_1_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_0_output, cuda_layer_0_output, (BATCH_SIZE*32*32*3*sizeof(unsigned char)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_1_bias, layer_1_bias, (128*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_1_weight, cuda_layer_1_weight, (3*3*3*128*sizeof(signed char)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 32;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 32;
    const int GRIDZSIZE = 128;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);
    
    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer1_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_0_output, d_layer_1_bias, d_cuda_layer_1_weight, d_cuda_layer_1_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_1_output, d_cuda_layer_1_output, (BATCH_SIZE*32*32*128*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_0_output);
    hipFree(d_layer_1_bias);
    hipFree(d_cuda_layer_1_weight);
    hipFree(d_cuda_layer_1_output);
    cudaCheckErrors("hipFree fail");

    // float sum = 0;
    // ofstream g("layer1/par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*32*32*128;i<(b+1)*32*32*128;i++){
    //         sum += cuda_layer_1_output[i];
    //         g<<cuda_layer_1_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }

    return milliseconds;
}