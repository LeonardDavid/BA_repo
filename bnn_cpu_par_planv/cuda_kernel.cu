#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "cuda_kernel.h"
#include "netW.hpp"
#include "utils.cuh"

using namespace std;

// Layer 4 - Convolution (xyz)

__global__ void layer4_conv_kernel(unsigned long long *d_cuda_layer_3_output, float *d_layer_4_bias, unsigned long long *d_cuda_layer_4_weight, signed short *d_cuda_layer_4_output){
    
    int N = 14, kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    int b = blockIdx.x; //batches in x-dir
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.x + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b<BATCH_SIZE){
            if(m<NR_NEURONS) {
                d_cuda_layer_4_output[index4D_cuda(b,h,w,m,14,14,64)] = d_layer_4_bias[m]; // = 0;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH<kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 14) {
                for (int kW = 0; kW<kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 14) {
                        if(b<BATCH_SIZE){
                            if(m<NR_NEURONS) {
                                for (int c = 0; c < 1; c++) {
                                    // atomicAddShort(&d_cuda_layer_4_output[index3D_cuda(h,w,m,14,64)], 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_4_weight[index4D_cuda(kH,kW,m,c,3,64,1)] ^ d_cuda_layer_3_output[index3D_cuda(iH,iW,c,14,64)])) - 64);
                                    d_cuda_layer_4_output[index4D_cuda(b,h,w,m,14,14,64)] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_4_weight[index4D_cuda(kH,kW,m,c,3,64,1)] ^ d_cuda_layer_3_output[index4D_cuda(b,iH,iW,c,14,14,64)])) - 64;
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

float layer4_conv_cuda(unsigned long long * cuda_layer_3_output, signed short * cuda_layer_4_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_4_weight
    unsigned long long *cuda_layer_4_weight = (unsigned long long *) layer_4_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_3_output; // storage on device for cuda_layer_3_output
    float *d_layer_4_bias; // storage on device for layer_4_bias
    unsigned long long *d_cuda_layer_4_weight; // storage on device for cuda_layer_4_weight
    signed short *d_cuda_layer_4_output; // RESULT storage on device for cuda_layer_4_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_3_output, BATCH_SIZE*12544*sizeof(unsigned long long)); // 196=14x14 dim of cuda_layer_4_output
    hipMalloc((void **) &d_layer_4_bias, 64*sizeof(float)); // 64 = dim of layer_4_bias
    hipMalloc((void **) &d_cuda_layer_4_weight, 36864*sizeof(unsigned long long)); // 576 = 3x3x64x[1x64] dim of layer_4_weight [ULL]
    hipMalloc((void **) &d_cuda_layer_4_output, BATCH_SIZE*12544*sizeof(signed short)); // 12544 = 14x14x64 dim of layer_4_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_3_output, cuda_layer_3_output, (BATCH_SIZE*12544*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_4_bias, layer_4_bias, (64*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_4_weight, cuda_layer_4_weight, (36864*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 14;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 14;
    const int GRIDZSIZE = NR_NEURONS;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE); // the 2 for loops 14 iterations each
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer4_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_3_output, d_layer_4_bias, d_cuda_layer_4_weight, d_cuda_layer_4_output);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // copy result from device to host
    hipMemcpy(cuda_layer_4_output, d_cuda_layer_4_output, (BATCH_SIZE*12544*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");    
    hipEventElapsedTime(&milliseconds, start, stop);

    // free the memory
    hipFree(d_cuda_layer_3_output);
    hipFree(d_layer_4_bias);
    hipFree(d_cuda_layer_4_weight);
    hipFree(d_cuda_layer_4_output);
    cudaCheckErrors("hipFree fail");

    // checksum L4 = 6334.000000
    // float sum = 0;
    // ofstream g("layer_4_par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*12544;i<(b+1)*12544;i++){
    //         sum += cuda_layer_4_output[i];
    //         g<<cuda_layer_4_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }

    return milliseconds;
}

// Layer 8 - Gemm (xyz)
__global__ void layer8_gemm_kernel(unsigned long long *d_cuda_layer_7_output, float *d_layer_8_bias, unsigned long long *d_cuda_layer_8_weight, signed short *d_cuda_layer_8_output){

    int z = blockDim.x * blockIdx.z + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int d = z*blockDim.x+y;

    int b = blockIdx.x;

    if(d < 2048){
        if(b < BATCH_SIZE){
            d_cuda_layer_8_output[b*2048 + d] = d_layer_8_bias[d];
            for (int i = 0; i < 49; i++) {
                d_cuda_layer_8_output[b*2048 + d] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_8_weight[d*49+i] ^ d_cuda_layer_7_output[i])) - 64;
            }
        }
    }
}

float layer8_gemm_cuda(unsigned long long * cuda_layer_7_output, signed short * cuda_layer_8_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_8_weight
    unsigned long long *cuda_layer_8_weight = (unsigned long long *) layer_8_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_7_output; // storage on device for cuda_layer_7_output
    float *d_layer_8_bias;  // storage on device for layer_8_bias
    unsigned long long *d_cuda_layer_8_weight; // storage on device for cuda_layer_8_weight
    signed short *d_cuda_layer_8_output; // RESULT storage on device for cuda_layer_8_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_7_output, BATCH_SIZE*49*sizeof(unsigned long long)); // 49=7x7 dim of cuda_layer_7_output
    hipMalloc((void **) &d_layer_8_bias, 2048*sizeof(float)); // 2048 = dim of layer_8_bias
    hipMalloc((void **) &d_cuda_layer_8_weight, 100352*sizeof(unsigned long long)); // 100352 = 2048x49 dim of layer_8_weight [ULL]
    hipMalloc((void **) &d_cuda_layer_8_output, BATCH_SIZE*2048*sizeof(signed short)); // 2048 = dim of layer_8_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_7_output, cuda_layer_7_output, (BATCH_SIZE*49*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_8_bias, layer_8_bias, (2048*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_8_weight, cuda_layer_8_weight, (100352*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 32;
    const int BLKYSIZE = 32;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 1;
    const int GRIDZSIZE = 2;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE); // 1 for loop 2048 iterations
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer8_gemm_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_7_output, d_layer_8_bias, d_cuda_layer_8_weight, d_cuda_layer_8_output);
    hipEventRecord(stop);
    cudaCheckErrors("Kernel launch failure");

    // copy result from device to host
    hipMemcpy(cuda_layer_8_output, d_cuda_layer_8_output, (BATCH_SIZE*2048*sizeof(signed short)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // free the memory
    hipFree(d_cuda_layer_7_output);
    hipFree(d_layer_8_bias);
    hipFree(d_cuda_layer_8_weight);
    hipFree(d_cuda_layer_8_output);
    cudaCheckErrors("hipFree fail");

    // checksum L8 = 8936.000000
    // float sum = 0;
    // ofstream g("layer_8_par1.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum=0;
    //     for(int i=b*2048;i<(b+1)*2048;i++){
    //         sum += cuda_layer_8_output[i];
    //         g<<cuda_layer_8_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"batch "<<b<<": "<<sum<<endl;
    // }
    return milliseconds;
}
