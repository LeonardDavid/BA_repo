#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <tuple>
#include <chrono>

#include "cuda_kernel.h"
#include "netW.hpp"
#include "utils.cuh"

using namespace std;

__global__ void layer1_conv_kernel(unsigned char *d_cuda_layer_0_output, float *d_layer_1_bias, signed char *d_cuda_layer_1_weight, float *d_cuda_layer_1_output){

    // https://github.com/ULHPC/tutorials/blob/devel/cuda/exercises/convolution/LoG_gpu_solution.cu

    int N = 32, kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.x + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < BATCH_SIZE){
            if(m < 128) {
                d_cuda_layer_1_output[index4D_cuda(b,h,w,m,32,32,128)] = d_layer_1_bias[m];
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 32) {
                for (int kW = 0; kW < kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 32) {
                        if(b < BATCH_SIZE){
                            for (int c = 0; c < 3; c++) {
                                if(m < 128) {
                                    d_cuda_layer_1_output[index4D_cuda(b,h,w,m,32,32,128)] += d_cuda_layer_1_weight[index4D_cuda(kH,kW,c,m,3,3,128)] * d_cuda_layer_0_output[index4D_cuda(b,iH,iW,c,32,32,3)];
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

float layer1_conv_cuda(unsigned char x[][32][32][3], float * cuda_layer_1_output){ // unsigned char * const x / unsigned char x[][32][32][3]
    
    setUniGPU();// use the second GPU on Uni-server because the first is used most of the time

    // initialize layer_0_output where x is the input image
    unsigned char (*layer_0_output)[BATCH_SIZE][32][32][3] = (unsigned char (*)[BATCH_SIZE][32][32][3]) x;

    // flatten 3D -> 1D arrays
    // flatten layer_1_weight
    signed char *cuda_layer_1_weight = (signed char *) layer_1_weight;

    // flatten layer_0_output
    unsigned char *cuda_layer_0_output = (unsigned char *) layer_0_output;

    // prepare for kernel call
    // declare storage on device
    unsigned char *d_cuda_layer_0_output; // storage on device for cuda_layer_0_output
    float *d_layer_1_bias; // storage on device for layer_1_bias
    signed char *d_cuda_layer_1_weight; // storage on device for cuda_layer_1_weight
    float *d_cuda_layer_1_output; // RESULT storage on device for cuda_layer_1_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_0_output, BATCH_SIZE*32*32*3*sizeof(unsigned char)); // 3072 = 32x32x3 dim of cuda_layer_0_output
    hipMalloc((void **) &d_layer_1_bias, 128*sizeof(float)); // 128 = dim of layer_1_bias
    hipMalloc((void **) &d_cuda_layer_1_weight, 3*3*3*128*sizeof(signed char)); // 3456 = 3x3x3x128 dim of layer_1_weight
    hipMalloc((void **) &d_cuda_layer_1_output, BATCH_SIZE*32*32*128*sizeof(float)); // 131072 = 32x32x128 dim of layer_1_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_0_output, cuda_layer_0_output, (BATCH_SIZE*32*32*3*sizeof(unsigned char)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_1_bias, layer_1_bias, (128*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_1_weight, cuda_layer_1_weight, (3*3*3*128*sizeof(signed char)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 32;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 32;
    const int GRIDZSIZE = 128;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);
    
    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer1_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_0_output, d_layer_1_bias, d_cuda_layer_1_weight, d_cuda_layer_1_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_1_output, d_cuda_layer_1_output, (BATCH_SIZE*32*32*128*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_0_output);
    hipFree(d_layer_1_bias);
    hipFree(d_cuda_layer_1_weight);
    hipFree(d_cuda_layer_1_output);
    cudaCheckErrors("hipFree fail");

    // // checksum L1 = 5720315.5
    // float sum_gpu = 0;
    // ofstream gg1("layer1/par.out");
    // for(int b=0;b<BATCH_SIZE;b++){
    //     sum_gpu = 0;
    //     for(int i=b*32*32*128;i<(b+1)*32*32*128;i++){
    //         sum_gpu += cuda_layer_1_output[i];
    //         gg1<<cuda_layer_1_output[i]<<" ";  
    //     }
    //     cout<<fixed<<"layer 1(GPU): batch "<<b<<": "<<sum_gpu<<endl;
    // }
    
    return milliseconds;
}

__global__ void layer3_conv_kernel(unsigned long long *d_cuda_layer_2_output, float *d_layer_3_bias, unsigned long long *d_cuda_layer_3_weight, float *d_cuda_layer_3_output){

    int N = 32, kernel_size = 3;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int m = blockIdx.z; // neurons in z-dir

    // batches in x-dir
    int b = blockIdx.x;
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.x + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < BATCH_SIZE){
            if(m < 128) {
                d_cuda_layer_3_output[index4D_cuda(b,h,w,m,32,32,128)] = d_layer_3_bias[m];
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < kernel_size; kH++){
            int iH = h * 1 + kH - 1;
            if (iH >= 0 && iH < 32) {
                for (int kW = 0; kW < kernel_size; kW++){
                    int iW = w * 1 + kW - 1;
                    if (iW >= 0 && iW < 32) {
                        if(b < BATCH_SIZE){
                            for (int c = 0; c < 2; c++) {
                                if(m < 128) {
                                    d_cuda_layer_3_output[index4D_cuda(b,h,w,m,32,32,128)] += 2 * __popcll((unsigned long long)~(unsigned long long)(d_cuda_layer_3_weight[index4D_cuda(kH,kW,m,c,3,128,2)] ^ d_cuda_layer_2_output[index4D_cuda(b,iH,iW,c,32,32,128)])) - 64; // ,128?
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

float layer3_conv_cuda(unsigned long long * cuda_layer_2_output, float * cuda_layer_3_output){
    
    setUniGPU();// use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // flatten layer_3_weight
    signed char *cuda_layer_3_weight = (signed char *) layer_3_weight;

    // prepare for kernel call
    // declare storage on device
    unsigned long long *d_cuda_layer_2_output; // storage on device for cuda_layer_2_output
    float *d_layer_3_bias; // storage on device for layer_3_bias
    unsigned long long *d_cuda_layer_3_weight; // storage on device for cuda_layer_3_weight
    float *d_cuda_layer_3_output; // RESULT storage on device for cuda_layer_3_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_2_output, BATCH_SIZE*32*32*2*64*sizeof(unsigned long long)); // 131072 = 32x32x2x64 dim of cuda_layer_2_output
    hipMalloc((void **) &d_layer_3_bias, 128*sizeof(float)); // 128 = dim of layer_3_bias
    hipMalloc((void **) &d_cuda_layer_3_weight, 3*3*128*2*64*sizeof(unsigned long long)); // 147456 = 3x3x128x2x64 dim of layer_3_weight
    hipMalloc((void **) &d_cuda_layer_3_output, BATCH_SIZE*32*32*128*sizeof(float)); // 131072 = 32x32x128 dim of layer_3_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_2_output, cuda_layer_2_output, (BATCH_SIZE*32*32*2*64*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    hipMemcpy(d_layer_3_bias, layer_3_bias, (128*sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(d_cuda_layer_3_weight, cuda_layer_3_weight, (3*3*128*2*64*sizeof(unsigned long long)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 32;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 32;
    const int GRIDZSIZE = 128;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);
    
    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer3_conv_kernel<<<numBlocks,threadsPerBlock>>>(d_cuda_layer_2_output, d_layer_3_bias, d_cuda_layer_3_weight, d_cuda_layer_3_output);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_3_output, d_cuda_layer_3_output, (BATCH_SIZE*32*32*128*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // free the memory
    hipFree(d_cuda_layer_2_output);
    hipFree(d_layer_3_bias);
    hipFree(d_cuda_layer_3_weight);
    hipFree(d_cuda_layer_3_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}

__global__ void layer4_maxpool_kernel(float *d_cuda_layer_3_output, float *d_cuda_layer_4_output, float lowest){

    int N = 16, kernel_size = 2;

    int tid = threadIdx.x; // = h
    int bid = blockIdx.y;  // = w
    int h = tid, w = bid;

    int c = blockIdx.z; // neurons in z-dir

    int b = blockIdx.x; // Batches index in grid x dir
    //each block is assigned to a row of an image, iy index of y value                  
    int iy = blockIdx.x + (kernel_size - 1)/2;  
    //each thread is assigned to a pixel of a row, ix index of x value
    int ix = threadIdx.x + (kernel_size - 1)/2; 
    
    //idx global index (all blocks) of the image pixel 
    int idx = iy*N +ix;

    // bias is applied to every pixel
    if(tid < N){
        if(b < BATCH_SIZE){
            if(c < 128) {
                d_cuda_layer_4_output[index4D_cuda(b,h,w,c,16,16,128)] = lowest;
            }
        }
    }

    __syncthreads();

    // edge pixels are skipped here because they cannot fit entire convolution window
    if(idx < N*N){
        for (int kH = 0; kH < kernel_size; kH++){
            for (int kW = 0; kW < kernel_size; kW++){
                if(b < BATCH_SIZE){
                    if(c < 128) {
                        d_cuda_layer_4_output[index4D_cuda(b,h,w,c,16,16,128)] = fmax(d_cuda_layer_3_output[index4D_cuda(b,(h * 2 + kH),(w * 2 + kW),c,32,32,128)], d_cuda_layer_4_output[index4D_cuda(b,h,w,c,16,16,128)]);
                    }
                }
            }
        }
    }
}

float layer4_maxpool_cuda(float * cuda_layer_3_output, float * cuda_layer_4_output){

    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time

    // flatten 3D -> 1D arrays
    // no 3D arrays to be flattened

    // prepare for kernel call
    // declare storage on device
    float *d_cuda_layer_3_output; // storage on device for cuda_layer_3_output
    float *d_cuda_layer_4_output; // RESULT storage on device for cuda_layer_4_output

    // allocate GPU device buffers
    hipMalloc((void **) &d_cuda_layer_3_output, BATCH_SIZE*32*32*128*sizeof(float)); // 131072 = 32x32x128 dim of layer_3_output
    hipMalloc((void **) &d_cuda_layer_4_output, BATCH_SIZE*16*16*128*sizeof(float)); // 32768 = 16x16x128 dim of layer_4_output
    cudaCheckErrors("Failed to allocate device buffer");

    // copy input data from host on device
    hipMemcpy(d_cuda_layer_3_output, cuda_layer_3_output, (BATCH_SIZE*32*32*128*sizeof(float)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    // define thread and block sizes
    const int BLKXSIZE = 16;
    const int BLKYSIZE = 1;
    const int GRIDXSIZE = BATCH_SIZE;
    const int GRIDYSIZE = 16;
    const int GRIDZSIZE = 128;

    const dim3 threadsPerBlock(BLKXSIZE, BLKYSIZE);
    const dim3 numBlocks(GRIDXSIZE, GRIDYSIZE, GRIDZSIZE);

    // std library not allowed on device
    const float LOWEST = std::numeric_limits<float>::lowest();

    // timing of the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;

    // compute result - kernel call
    hipEventRecord(start);
    layer4_maxpool_kernel<<<numBlocks, threadsPerBlock>>>(d_cuda_layer_3_output, d_cuda_layer_4_output, LOWEST);
    cudaCheckErrors("Kernel launch failure");
    hipEventRecord(stop);

    // copy result from device to host
    hipMemcpy(cuda_layer_4_output, d_cuda_layer_4_output, (BATCH_SIZE*16*16*128*sizeof(float)), hipMemcpyDeviceToHost);
    cudaCheckErrors("CUDA memcpy failure");

    // synchronize threads
    hipDeviceSynchronize();
    cudaCheckErrors("CUDA synchronize failure");
    hipEventElapsedTime(&milliseconds, start, stop);

    // free the memory
    hipFree(d_cuda_layer_3_output);
    hipFree(d_cuda_layer_4_output);
    cudaCheckErrors("hipFree fail");

    return milliseconds;
}

float layer6_conv_cuda(unsigned long long * cuda_layer_5_output, float * cuda_layer_6_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer8_conv_cuda(unsigned long long * cuda_layer_7_output, float * cuda_layer_8_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer9_maxpool_cuda(float * cuda_layer_8_output, float * cuda_layer_9_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer11_conv_cuda(unsigned long long * cuda_layer_10_output, float * cuda_layer_11_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer13_conv_cuda(unsigned long long * cuda_layer_12_output, float * cuda_layer_13_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer14_maxpool_cuda(float * cuda_layer_13_output, float * cuda_layer_14_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer17_gemm_cuda(unsigned long long * cuda_layer_16_output, float * cuda_layer_17_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}

float layer19_gemm_cuda(unsigned long long * cuda_layer_18_output, float * cuda_layer_19_output){
    
    setUniGPU(); // use the second GPU on Uni-server because the first is used most of the time
    
    return 0;
}
